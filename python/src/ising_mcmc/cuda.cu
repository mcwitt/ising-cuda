#include <hiprand.h>
#include <format>
#include <nanobind/nanobind.h>
#include <nanobind/ndarray.h>
#include <nanobind/stl/tuple.h>
#include <stdexcept>
#include <vector>

#include "twod.cuh"

namespace nb = nanobind;

NB_MODULE(cuda, m) {
  m.def(
      "sweeps",
      [](const nb::ndarray<int, nb::device::cpu> &spin,
         const nb::ndarray<float, nb::device::cpu> &hext,
         const nb::ndarray<float, nb::ndim<1>, nb::device::cpu> &temps,
         const unsigned int n_sweeps,
         const unsigned long seed) {
        if (spin.ndim() < 2) {
          throw std::invalid_argument("spin must have at minimum 2 dimensions");
        }

        const unsigned int nt = spin.shape(0);
        const unsigned int l = spin.shape(1);

        for (auto i = 1u; i < spin.ndim(); ++i) {
          if (spin.shape(i) != l)
            throw std::invalid_argument(std::format(
                "only hypercubic lattices are supported, but got conflicting "
                "dimensions: spin.shape(1) = {}; spin.shape({}) = {}",
                l,
                i,
                spin.shape(i)));
        }

        if (hext.ndim() != spin.ndim()) {
          throw std::invalid_argument(std::format(
              "spin and hext must have same shape, but got conflicting numbers "
              "of dimensions {} and {}",
              spin.ndim(),
              hext.ndim()));
        }

        for (auto i = 0u; i < spin.ndim(); ++i) {
          if (spin.shape(i) != hext.shape(i)) {
            throw std::invalid_argument(std::format(
                "spin and hext must have same shape, but got conflicting sizes "
                "{} and {} in dimension {}",
                spin.shape(i),
                hext.shape(i),
                i));
          }
        }

        if (temps.size() != nt)
          throw std::invalid_argument(std::format(
              "first dimensions of spin, hext, and temps must match, but got "
              "{} and {}",
              spin.shape(0),
              hext.shape(0),
              temps.size()));

        for (auto i = 0u; i < spin.size(); ++i) {
          const auto s = spin.data()[i];
          if (s != 1 && s != -1) {
            throw std::invalid_argument(
                std::format("invalid value in spin: {}", s));
          }
        }

        int *d_spin;
        float *d_hext;
        float *d_noise;
        float *d_temps;
        unsigned long long *d_naccept;
        int *d_spinsum;
        float *d_m2sum;
        float *d_m4sum;

        hipMalloc(&d_spin, nt * l * l * sizeof(int));
        hipMemcpy(
            d_spin,
            spin.data(),
            nt * l * l * sizeof(int),
            hipMemcpyHostToDevice);

        hipMalloc(&d_hext, nt * l * l * sizeof(float));
        hipMemcpy(
            d_hext,
            hext.data(),
            nt * l * l * sizeof(float),
            hipMemcpyHostToDevice);

        hipMalloc(&d_temps, nt * sizeof(float));
        hipMemcpy(
            d_temps, temps.data(), nt * sizeof(float), hipMemcpyHostToDevice);

        hipMalloc(&d_noise, nt * l * l * sizeof(float));
        hipMalloc(&d_naccept, nt * sizeof(unsigned long long));
        hipMalloc(&d_spinsum, nt * sizeof(int));
        hipMalloc(&d_m2sum, nt * sizeof(float));
        hipMalloc(&d_m4sum, nt * sizeof(float));

        constexpr dim3 blockDim(32, 32, 1);
        dim3 gridDim(
            ceil_div(l, blockDim.x),
            ceil_div(l, blockDim.y),
            ceil_div(nt, blockDim.z));

        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, seed);
        hiprandGenerateUniform(gen, d_noise, nt * l * l);

        hipMemset(d_naccept, 0, nt * sizeof(unsigned long long));

        hipMemset(d_m2sum, 0, nt * sizeof(float));
        hipMemset(d_m4sum, 0, nt * sizeof(float));

        for (unsigned int isweep = 0; isweep < n_sweeps; ++isweep) {
          hiprandGenerateUniform(gen, d_noise, nt * l * l);

          // checkerboard updates

          static_assert(blockDim.z == 1, "require blockDim.z == 1");

          k_sweep<<<gridDim, blockDim>>>(
              0,
              l,
              d_hext,
              nt,
              d_temps,
              d_noise,
              d_spin,
              d_naccept); // light squares

          k_sweep<<<gridDim, blockDim>>>(
              1,
              l,
              d_hext,
              nt,
              d_temps,
              d_noise,
              d_spin,
              d_naccept); // dark squares

          // accumulate magnetization

          hipMemset(d_spinsum, 0, nt * sizeof(int));

          k_accum<<<dim3(ceil_div(l * l, 32), nt, 1), dim3(32, 1, 1)>>>(
              l * l, nt, d_spin, d_spinsum);

          k_accum_scalar_moments<<<ceil_div(nt, 32), 32>>>(
              l * l, nt, d_spinsum, d_m2sum, d_m4sum);
        }
        std::vector<int> spin_(nt * l * l);
        hipMemcpy(
            spin_.data(),
            d_spin,
            nt * l * l * sizeof(int),
            hipMemcpyDeviceToHost);

        std::vector<unsigned long long> naccept(nt);
        hipMemcpy(
            naccept.data(),
            d_naccept,
            nt * sizeof(unsigned long long),
            hipMemcpyDeviceToHost);

        std::vector<float> m2avg(nt);
        hipMemcpy(
            m2avg.data(), d_m2sum, nt * sizeof(float), hipMemcpyDeviceToHost);

        std::vector<float> m4avg(nt);
        hipMemcpy(
            m4avg.data(), d_m4sum, nt * sizeof(float), hipMemcpyDeviceToHost);

        std::vector<float> acceptrate(nt);
        for (unsigned int t = 0; t < nt; ++t) {
          acceptrate[t] = (float)naccept[t] / n_sweeps / l / l;
          m2avg[t] /= n_sweeps;
          m4avg[t] /= n_sweeps;
        }

        hipFree(d_temps);
        hipFree(d_spin);
        hipFree(d_noise);
        hipFree(d_naccept);
        hipFree(d_spinsum);
        hipFree(d_m2sum);
        hipFree(d_m4sum);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
          fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
          throw std::runtime_error(hipGetErrorString(err));
        }

        return std::make_tuple(
            nb::ndarray<nb::numpy, int, nb::ndim<3>>(spin_.data(), {nt, l, l})
                .cast(),
            nb::ndarray<nb::numpy, float, nb::ndim<1>>(acceptrate.data(), {nt})
                .cast(),
            nb::ndarray<nb::numpy, float, nb::ndim<1>>(m2avg.data(), {nt})
                .cast(),
            nb::ndarray<nb::numpy, float, nb::ndim<1>>(m4avg.data(), {nt})
                .cast());
      });
}
