#include "hip/hip_runtime.h"
#include "twod.cuh"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand.h>

__global__ void k_sweep(
    const unsigned int parity,
    const unsigned int l,
    const float *hext,
    const unsigned int nt,
    const float *temps,
    const float *__restrict__ noise,
    int *const __restrict__ spin,
    unsigned long long *const __restrict__ naccept) {

  const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
  const unsigned int t = blockIdx.z * blockDim.z + threadIdx.z;

  if (t >= nt || i >= l || j >= l)
    return;

  int local_naccept = 0;

  if ((i + j) % 2 == parity) {

    const unsigned int iprev = (i == 0) ? l - 1 : i - 1;
    const unsigned int jprev = (j == 0) ? l - 1 : j - 1;

    const unsigned int inext = (i == l - 1) ? 0 : i + 1;
    const unsigned int jnext = (j == l - 1) ? 0 : j + 1;

    const unsigned int offset = t * l * l;

    const int nbrsum =
        spin[offset + i * l + jprev] + spin[offset + i * l + jnext] +
        spin[offset + iprev * l + j] + spin[offset + inext * l + j];

    const float h = (float)nbrsum + hext[offset + i * l + j];
    const unsigned int idx = offset + i * l + j;
    const int s = spin[idx];
    const float de = 2.0f * (float)s * h;

    if (de <= 0) {
      spin[idx] = -s;
      local_naccept = 1;
    } else {
      const float temp = temps[t];
      const float prob = exp(-static_cast<float>(de) / temp);
      if (noise[idx] < prob) {
        spin[idx] = -s;
        local_naccept = 1;
      }
    }
  }

  k_accum_block_sum(local_naccept, &naccept[t]);
}

__global__ void k_accum_scalar_moments(
    const unsigned int n,
    const unsigned int nt,
    const int *const __restrict__ sum,
    float *const __restrict__ m2sum,
    float *const __restrict__ m4sum) {

  const unsigned int t = blockIdx.x * blockDim.x + threadIdx.x;

  if (t >= nt)
    return;

  float m = static_cast<float>(sum[t]) / static_cast<float>(n);

  atomicAdd(&m2sum[t], m * m);
  atomicAdd(&m4sum[t], m * m * m * m);
}
