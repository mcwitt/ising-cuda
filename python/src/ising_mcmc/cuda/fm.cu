#include "hip/hip_runtime.h"
#include <format>
#include <span>
#include <vector>

#include <hiprand.h>

#include "fm.cuh"
#include "fm_2d.cuh"
#include "fm_nd.cuh"
#include "hypercube.hpp"

template <typename T>
__global__ void k_accum(
    const unsigned int n,
    const size_t nt,
    const T *const __restrict__ vals,
    T *const __restrict__ out) {

  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int t = blockIdx.y * blockDim.y + threadIdx.y;

  if (t >= nt || i >= n)
    return;

  int local_sum = vals[t * n + i];

  k_accum_block_sum(local_sum, &out[t]);
}

using SweepKernel = void (*)(
    const unsigned int,
    const unsigned int *const __restrict__,
    const float *const __restrict__,
    const size_t,
    const float *const __restrict__,
    const float *const __restrict__,
    int *const __restrict__,
    unsigned long long *const __restrict__);

auto get_hypercube_sweep_kernel(const unsigned int d) -> SweepKernel {
  using ising_mcmc::cuda::fm::k_sweep_nd;
  switch (d) {
  case 1:
    return k_sweep_nd<1>;
  case 2:
    return k_sweep_nd<2>;
  case 3:
    return k_sweep_nd<3>;
  case 4:
    return k_sweep_nd<4>;
  case 5:
    return k_sweep_nd<5>;
  case 6:
    return k_sweep_nd<6>;
  case 7:
    return k_sweep_nd<7>;
  case 8:
    return k_sweep_nd<8>;
  case 9:
    return k_sweep_nd<9>;
  case 10:
    return k_sweep_nd<10>;
  default:
    throw std::invalid_argument(
        std::format(
            "number of dimensions must be between 1 and 10, but got {}", d));
  }
}

auto get_sweep_kernel_and_launch_params(
    const unsigned int nt,
    const unsigned int d,
    const unsigned int l,
    const unsigned int n) -> std::tuple<SweepKernel, dim3, dim3> {
  switch (d) {
  case 2:
    using ising_mcmc::cuda::fm::TILE_SIZE;
    return std::make_tuple(
        ising_mcmc::cuda::fm::k_sweep_2d,
        dim3(TILE_SIZE, TILE_SIZE, 1),
        dim3(ceil_div(l, TILE_SIZE), ceil_div(l, TILE_SIZE), nt));
  default:
    return std::make_tuple(
        get_hypercube_sweep_kernel(d), dim3(TPB), dim3(ceil_div(n, TPB), nt));
  }
}

auto ising_mcmc::cuda::fm::sweeps(
    const unsigned int d,
    const unsigned int l,
    const std::span<const float> hext,
    const std::span<const float> temps,
    const unsigned int n_sweeps,
    const unsigned long seed,
    std::span<int> spin,
    std::span<double> acceptrate,
    std::span<double> m2,
    std::span<double> m4) -> void {

  const auto strides = compute_strides(d, l);
  const unsigned int n = strides[d];
  const unsigned int nt = temps.size();

  unsigned int *d_strides;
  int *d_spin;
  float *d_hext;
  float *d_noise;
  float *d_temps;
  unsigned long long *d_naccept;
  int *d_spinsum;

  hipMalloc(&d_strides, (d + 1) * sizeof(unsigned int));
  hipMemcpy(
      d_strides,
      strides.data(),
      (d + 1) * sizeof(unsigned int),
      hipMemcpyHostToDevice);

  hipMalloc(&d_spin, nt * n * sizeof(int));
  hipMemcpy(d_spin, spin.data(), nt * n * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc(&d_hext, nt * n * sizeof(float));
  hipMemcpy(
      d_hext, hext.data(), nt * n * sizeof(float), hipMemcpyHostToDevice);

  hipMalloc(&d_temps, nt * sizeof(float));
  hipMemcpy(d_temps, temps.data(), nt * sizeof(float), hipMemcpyHostToDevice);

  hipMalloc(&d_noise, nt * n * sizeof(float));
  hipMalloc(&d_naccept, nt * sizeof(unsigned long long));
  hipMalloc(&d_spinsum, nt * sizeof(int));

  auto [k_sweep, block_dim, grid_dim] =
      get_sweep_kernel_and_launch_params(nt, d, l, n);

  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);

  hipMemset(d_naccept, 0, nt * sizeof(unsigned long long));

  for (auto isweep = 0u; isweep < n_sweeps; ++isweep) {
    hiprandGenerateUniform(gen, d_noise, nt * n);

    // checkerboard updates

    k_sweep<<<grid_dim, block_dim>>>(
        0,
        d_strides,
        d_hext,
        nt,
        d_temps,
        d_noise,
        d_spin,
        d_naccept); // light squares

    k_sweep<<<grid_dim, block_dim>>>(
        1,
        d_strides,
        d_hext,
        nt,
        d_temps,
        d_noise,
        d_spin,
        d_naccept); // dark squares

    // accumulate magnetization

    hipMemset(d_spinsum, 0, nt * sizeof(int));

    k_accum<<<dim3(ceil_div(n, 256), nt), dim3(256)>>>(
        n, nt, d_spin, d_spinsum);

    std::vector<int> spinsum(nt);
    hipMemcpy(
        spinsum.data(), d_spinsum, nt * sizeof(int), hipMemcpyDeviceToHost);
    for (auto t = 0u; t < nt; ++t) {
      double m = spinsum[t] / static_cast<double>(n);
      double m2_ = m * m;
      double m4_ = m2_ * m2_;
      m2[t] += m2_;
      m4[t] += m4_;
    }
  }

  hipMemcpy(spin.data(), d_spin, nt * n * sizeof(int), hipMemcpyDeviceToHost);

  std::vector<unsigned long long> naccept(nt);
  hipMemcpy(
      naccept.data(),
      d_naccept,
      nt * sizeof(unsigned long long),
      hipMemcpyDeviceToHost);

  for (unsigned int t = 0; t < nt; ++t) {
    acceptrate[t] = static_cast<double>(naccept[t]) / n_sweeps / n;
    m2[t] /= n_sweeps;
    m4[t] /= n_sweeps;
  }

  hipFree(d_strides);
  hipFree(d_temps);
  hipFree(d_spin);
  hipFree(d_hext);
  hipFree(d_noise);
  hipFree(d_naccept);
  hipFree(d_spinsum);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    throw std::runtime_error(hipGetErrorString(err));
  }
}
