#include "hip/hip_runtime.h"
#include "fm_2d.cuh"
#include "kernel_utils.cuh"

__global__ void ising_mcmc::cuda::fm::k_sweep_2d(
    const unsigned int parity,
    const unsigned int *const __restrict__ d_strides,
    const float *hext,
    const size_t nt,
    const float *temps,
    const float *__restrict__ noise,
    int *const __restrict__ spin,
    unsigned long long *const __restrict__ naccept) {

  const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
  const unsigned int t = blockIdx.z * blockDim.z + threadIdx.z;

  const unsigned int l = d_strides[1];

  if (t >= nt || i >= l || j >= l)
    return;

  int local_naccept = 0;

  if ((i + j) % 2 == parity) {

    const unsigned int iprev = (i == 0) ? l - 1 : i - 1;
    const unsigned int jprev = (j == 0) ? l - 1 : j - 1;

    const unsigned int inext = (i == l - 1) ? 0 : i + 1;
    const unsigned int jnext = (j == l - 1) ? 0 : j + 1;

    const unsigned int offset = t * l * l;

    const int nbrsum =
        spin[offset + i * l + jprev] + spin[offset + i * l + jnext] +
        spin[offset + iprev * l + j] + spin[offset + inext * l + j];

    const unsigned int idx = offset + i * l + j;
    const float h = static_cast<float>(nbrsum) + hext[idx];
    const int s = spin[idx];
    const float de = static_cast<float>(2 * s) * h;

    if (de <= 0) {
      spin[idx] = -s;
      local_naccept = 1;
    } else {
      const float temp = temps[t];
      const float prob = exp(-de / temp);
      if (noise[idx] < prob) {
        spin[idx] = -s;
        local_naccept = 1;
      }
    }
  }

  k_accum_block_sum(local_naccept, &naccept[t]);
}
