#include <array>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <vector>

constexpr unsigned int D_ = 2;

constexpr unsigned int WARP_SIZE = 32;

auto compute_strides(unsigned int l) -> std::array<unsigned int, D_ + 1> {
  std::array<unsigned int, D_ + 1> strides{};
  strides[0] = 1;
  for (int i = 1; i <= D_; ++i) {
    strides[i] = strides[i - 1] * l;
  }
  return strides;
}

__constant__ unsigned int c_strides[D_ + 1];

__global__ void k_init_random(
    const unsigned int n,
    const float *const __restrict__ noise,
    int *const __restrict__ spin) {
  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= n)
    return;

  const int p = noise[i] < 0.5;
  spin[i] = 2 * p - 1;
};

constexpr __host__ __device__ auto ceil_div(unsigned int x, unsigned int y)
    -> unsigned int {
  return (x + y - 1) / y;
}

template <typename T> __device__ void k_accum_block_sum(int &val, T *out) {

  /* Computes the sum of val for all threads in a block and stores the
    result in out. */

  // 1. Compute sum of values in each warp

  for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
    val += __shfl_down_sync(0xFFFFFFFF, val, offset);
  }

  // At this point, the first thread in each warp ("warp leader") has
  // for its value the sum of the values over the warp.

  // 2. Warp leaders store warp sums in shared memory

  __shared__ int warp_sums[WARP_SIZE];

  const unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x;

  if (tid % WARP_SIZE == 0) {
    warp_sums[tid / WARP_SIZE] = val;
  }

  // 3. Threads in first warp reduce warp sums

  __syncthreads(); // ensure all threads see the final value of warp_sums

  const unsigned int tpb = blockDim.x * blockDim.y;
  const unsigned int nwarps = ceil_div(tpb, WARP_SIZE);

  if (tid < WARP_SIZE) {
    val = (tid < nwarps) ? warp_sums[tid] : 0;

    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
      val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }

    // First warp leader updates out
    if (tid == 0) {
      atomicAdd(out, val);
    }
  }
}

__global__ void k_sweep(
    const unsigned int parity,
    const float hext,
    const size_t nt,
    const float *temps,
    const float *__restrict__ noise,
    int *const __restrict__ spin,
    unsigned long long *const __restrict__ naccept) {

  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  // reduction of naccept assumes block has consistent temperature
  assert(blockDim.z == 1);
  const unsigned int t = blockIdx.y;

  const unsigned int n = c_strides[D_];

  if (t >= nt || i >= n)
    return;

  int local_naccept = 0;

  unsigned int ccurr[D_];
  unsigned int rem = i;

  for (int d = D_ - 1; d >= 0; d--) {
    const unsigned int stride = c_strides[d];
    ccurr[d] = rem / stride;
    rem %= stride;
  }

  unsigned int dist = 0;
  for (unsigned int k : ccurr) {
    dist += k;
  }

  if (dist % 2 == parity) {
    const unsigned int l = c_strides[1];

    unsigned int cprev[D_];
    unsigned int cnext[D_];

    for (int d = 0; d < D_; ++d) {
      cprev[d] = (ccurr[d] == 0) ? l - 1 : ccurr[d] - 1;
      cnext[d] = (ccurr[d] == l - 1) ? 0 : ccurr[d] + 1;
    }

    const unsigned int offset = t * n;

    int nbrsum = 0;
    for (int d = 0; d < D_; ++d) {
      unsigned int iprev = 0;
      unsigned int inext = 0;

      // compute indices of forward and reverse neighbors in dimension d
      for (int dp = 0; dp < D_; ++dp) {
        iprev += c_strides[dp] * ((dp == d) ? cprev[dp] : ccurr[dp]);
        inext += c_strides[dp] * ((dp == d) ? cnext[dp] : ccurr[dp]);
      }

      nbrsum += spin[offset + iprev];
      nbrsum += spin[offset + inext];
    }

    const float h = static_cast<float>(nbrsum) + hext;
    const unsigned int idx = offset + i;
    const int s = spin[idx];
    const float de = static_cast<float>(2 * s) * h;

    if (de <= 0) {
      spin[idx] = -s;
      local_naccept = 1;
    } else {
      const float temp = temps[t];
      const float prob = exp(-de / temp);
      if (noise[idx] < prob) {
        spin[idx] = -s;
        local_naccept = 1;
      }
    }
  }

  k_accum_block_sum(local_naccept, &naccept[t]);
}

template <typename T>
__global__ void k_accum(
    const unsigned int n,
    const size_t nt,
    const T *const __restrict__ vals,
    T *const __restrict__ out) {

  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int t = blockIdx.y * blockDim.y + threadIdx.y;

  if (t >= nt || i >= n)
    return;

  int local_sum = vals[t * n + i];

  k_accum_block_sum(local_sum, &out[t]);
}

auto parse_float(const char *s) -> float {
  char *endptr = nullptr;
  float r = strtof(s, &endptr);
  if (*endptr != '\0') {
    fprintf(stderr, "Invalid float: %s\n", s);
    exit(1);
  }
  return r;
}

auto parse_long(const char *s) -> long {
  char *endptr = nullptr;
  long r = strtol(s, &endptr, 10);
  if (*endptr != '\0') {
    fprintf(stderr, "Invalid long: %s\n", s);
    exit(1);
  }
  return r;
}

void parse_args(
    int argc,
    char *argv[],
    unsigned int *l,
    float *hext,
    unsigned long *n_samples,
    unsigned long *sweeps_per_sample,
    unsigned long *seed) {
  if (argc != 6) {
    fprintf(
        stderr,
        "Usage: %s L H_EXT N_SAMPLES SWEEPS_PER_SAMPLE SEED\n",
        argv[0]);
    exit(1);
  }
  *l = parse_long(argv[1]);
  *hext = parse_float(argv[2]);
  *n_samples = parse_long(argv[3]);
  *sweeps_per_sample = parse_long(argv[4]);
  *seed = parse_long(argv[5]);
}

auto read_floats() -> std::vector<float> {
  std::vector<float> vals;
  float val;

  while (scanf("%f", &val) == 1) {
    vals.push_back(val);
  }

  return vals;
}

auto main(int argc, char *argv[]) -> int {
  unsigned int l;
  float hext;
  unsigned long n_samples;
  unsigned long sweeps_per_sample;
  unsigned long seed;
  parse_args(argc, argv, &l, &hext, &n_samples, &sweeps_per_sample, &seed);

  auto strides = compute_strides(l);
  const unsigned int n = strides[D_];

  const std::vector<float> temps = read_floats();
  const size_t nt = temps.size();

  int *d_spin;
  float *d_noise;
  float *d_temps;
  unsigned long long *d_naccept;
  int *d_spinsum;

  hipMalloc(&d_spin, nt * n * sizeof(int));
  hipMalloc(&d_noise, nt * n * sizeof(float));

  hipMalloc(&d_temps, nt * sizeof(float));
  hipMalloc(&d_naccept, nt * sizeof(unsigned long long));
  hipMalloc(&d_spinsum, nt * sizeof(int));

  hipMemcpyToSymbol(
      HIP_SYMBOL(c_strides), strides.data(), (D_ + 1) * sizeof(unsigned int));

  hipMemcpy(d_temps, temps.data(), nt * sizeof(float), hipMemcpyHostToDevice);

  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);
  hiprandGenerateUniform(gen, d_noise, nt * n);

  k_init_random<<<ceil_div(nt * n, 256), 256>>>(nt * n, d_noise, d_spin);

  printf(
      "D,L,h_ext,sweeps_per_sample,seed,temperature,sample,accept_rate,"
      "<m^2>,<m^4>,time_s\n");

  for (int isample = 0; isample < n_samples; ++isample) {
    clock_t start_time = clock();
    std::vector<double> m2sum(nt);
    std::vector<double> m4sum(nt);

    hipMemset(d_naccept, 0, nt * sizeof(unsigned long long));

    for (int isweep = 0; isweep < sweeps_per_sample; ++isweep) {
      hiprandGenerateUniform(gen, d_noise, nt * n);

      // checkerboard updates

      constexpr dim3 blockDim(256);
      dim3 gridDim(ceil_div(n, blockDim.x), ceil_div(nt, blockDim.z), 1);

      static_assert(blockDim.z == 1, "require blockDim.z == 1");

      k_sweep<<<gridDim, blockDim>>>(
          0,
          hext,
          nt,
          d_temps,
          d_noise,
          d_spin,
          d_naccept); // light squares

      k_sweep<<<gridDim, blockDim>>>(
          1,
          hext,
          nt,
          d_temps,
          d_noise,
          d_spin,
          d_naccept); // dark squares

      // accumulate magnetization

      hipMemset(d_spinsum, 0, nt * sizeof(int));

      k_accum<<<dim3(ceil_div(n, 256), nt), dim3(256)>>>(
          n, nt, d_spin, d_spinsum);

      std::vector<int> spinsum(nt);
      hipMemcpy(
          spinsum.data(), d_spinsum, nt * sizeof(int), hipMemcpyDeviceToHost);
      for (int t = 0; t < nt; ++t) {
        double m = spinsum[t] / static_cast<double>(n);
        double m2 = m * m;
        double m4 = m2 * m2;
        m2sum[t] += m2;
        m4sum[t] += m4;
      }
    }

    std::vector<unsigned long long> naccept(nt);
    hipMemcpy(
        naccept.data(),
        d_naccept,
        nt * sizeof(unsigned long long),
        hipMemcpyDeviceToHost);

    clock_t end_time = clock();

    const double time_s = (double)(end_time - start_time) / CLOCKS_PER_SEC;

    for (int t = 0; t < nt; ++t) {
      const double accept_rate =
          (double)naccept[t] / (double)sweeps_per_sample / n;
      const double m2avg = m2sum[t] / (double)sweeps_per_sample;
      const double m4avg = m4sum[t] / (double)sweeps_per_sample;

      printf(
          "%u,%u,%g,%ld,%ld,%g,%d,%g,%g,%g,%g\n",
          D_,
          l,
          hext,
          sweeps_per_sample,
          seed,
          temps[t],
          isample,
          accept_rate,
          m2avg,
          m4avg,
          time_s);
    }
  }

  hipFree(d_temps);
  hipFree(d_spin);
  hipFree(d_noise);
  hipFree(d_naccept);
  hipFree(d_spinsum);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    return 1;
  }

  return 0;
}
