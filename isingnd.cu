#include <array>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <vector>

constexpr unsigned int D_ = 2;

#ifdef L
constexpr unsigned long L_ = L;
#else
constexpr unsigned long L_ = 64;
#endif

constexpr auto compute_strides() -> std::array<unsigned int, D_ + 1> {
  std::array<unsigned int, D_ + 1> strides{};
  strides[0] = 1;
  for (int i = 1; i <= D_; i++) {
    strides[i] = strides[i - 1] * L_;
  }
  return strides;
}

constexpr __constant__ auto strides = compute_strides();
constexpr int N = strides[D_];

__global__ void k_init_random(
    const size_t nt,
    const float *const __restrict__ noise,
    int *const __restrict__ spin) {
  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= nt * N)
    return;

  const int p = noise[i] < 0.5;
  spin[i] = 2 * p - 1;
};

constexpr __host__ __device__ auto ceil_div(unsigned int x, unsigned int y)
    -> unsigned int {
  return (x + y - 1) / y;
}

template <typename T> __device__ void k_accum_block_sum(int &val, T *out) {

  /* Computes the sum of val for all threads in a block and stores the
    result in out. */

  // 1. Compute sum of values in each warp

  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    val += __shfl_down_sync(0xFFFFFFFF, val, offset);
  }

  // At this point, the first thread in each warp ("warp leader") has
  // for its value the sum of the values over the warp.

  // 2. Warp leaders store warp sums in shared memory

  __shared__ int warp_sums[32];

  const unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x;

  if (tid % warpSize == 0) {
    warp_sums[tid / warpSize] = val;
  }

  // 3. Threads in first warp reduce warp sums

  __syncthreads(); // ensure all threads see the final value of warp_sums

  const unsigned int tpb = blockDim.x * blockDim.y;
  const unsigned int nwarps = ceil_div(tpb, warpSize);

  if (tid < warpSize) {
    val = (tid < nwarps) ? warp_sums[tid] : 0;

    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }

    // First warp leader updates out
    if (tid == 0) {
      atomicAdd(out, val);
    }
  }
}

__global__ void k_sweep(
    const unsigned int parity,
    const float hext,
    const size_t nt,
    const float *temps,
    const float *__restrict__ noise,
    int *const __restrict__ spin,
    unsigned long long *const __restrict__ naccept) {

  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int t = blockIdx.y * blockDim.y + threadIdx.y;

  if (t >= nt || i >= N)
    return;

  int local_naccept = 0;

  unsigned int ccurr[D_];
  unsigned int rem = i;

  for (int d = D_ - 1; d >= 0; d--) {
    const unsigned int stride = strides[d];
    ccurr[d] = rem / stride;
    rem %= stride;
  }

  unsigned int dist = 0;
  for (unsigned int k : ccurr) {
    dist += k;
  }

  if (dist % 2 == parity) {

    unsigned int cprev[D_];
    unsigned int cnext[D_];

    for (int d = 0; d < D_; d++) {
      cprev[d] = (ccurr[d] == 0) ? L_ - 1 : ccurr[d] - 1;
      cnext[d] = (ccurr[d] == L_ - 1) ? 0 : ccurr[d] + 1;
    }

    const unsigned int offset = t * N;

    int nbrsum = 0;
    for (int d = 0; d < D_; d++) {
      unsigned int iprev = 0;
      unsigned int inext = 0;

      // compute indices of forward and reverse neighbors in dimension d
      for (int dp = 0; dp < D_; dp++) {
        iprev += strides[dp] * ((dp == d) ? cprev[dp] : ccurr[dp]);
        inext += strides[dp] * ((dp == d) ? cnext[dp] : ccurr[dp]);
      }

      nbrsum += spin[offset + iprev];
      nbrsum += spin[offset + inext];
    }

    const float h = (float)nbrsum + hext;
    const unsigned int idx = offset + i;
    const int s = spin[idx];
    const float de = 2.0f * (float)s * h;

    if (de <= 0) {
      spin[idx] = -s;
      local_naccept = 1;
    } else {
      const float temp = temps[t];
      const float prob = exp(-static_cast<float>(de) / temp);
      if (noise[idx] < prob) {
        spin[idx] = -s;
        local_naccept = 1;
      }
    }
  }

  k_accum_block_sum(local_naccept, &naccept[t]);
}

template <typename T>
__global__ void k_accum(
    const size_t nt,
    const T *const __restrict__ vals,
    T *const __restrict__ out) {

  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int t = blockIdx.y * blockDim.y + threadIdx.y;

  if (t >= nt || i >= N)
    return;

  int local_sum = vals[t * N + i];

  k_accum_block_sum(local_sum, &out[t]);
}

__global__ void k_accum_scalar_moments(
    const size_t nt,
    const int *const __restrict__ sum,
    float *const __restrict__ m2sum,
    float *const __restrict__ m4sum) {

  const unsigned int t = blockIdx.x * blockDim.x + threadIdx.x;

  if (t >= nt)
    return;

  float m = static_cast<float>(sum[t]) / static_cast<float>(N);

  atomicAdd(&m2sum[t], m * m);
  atomicAdd(&m4sum[t], m * m * m * m);
}

auto parse_float(const char *s) -> float {
  char *endptr = nullptr;
  float r = strtof(s, &endptr);
  if (*endptr != '\0') {
    fprintf(stderr, "Invalid float: %s\n", s);
    exit(1);
  }
  return r;
}

auto parse_long(const char *s) -> long {
  char *endptr = nullptr;
  long r = strtol(s, &endptr, 10);
  if (*endptr != '\0') {
    fprintf(stderr, "Invalid long: %s\n", s);
    exit(1);
  }
  return r;
}

void parse_args(
    int argc,
    char *argv[],
    float *hext,
    long *n_samples,
    long *sweeps_per_sample,
    unsigned long *seed) {
  if (argc != 5) {
    fprintf(
        stderr, "Usage: %s H_EXT N_SAMPLES SWEEPS_PER_SAMPLE SEED\n", argv[0]);
    exit(1);
  }
  *hext = parse_float(argv[1]);
  *n_samples = parse_long(argv[2]);
  *sweeps_per_sample = parse_long(argv[3]);
  *seed = parse_long(argv[4]);
}

auto read_floats() -> std::vector<float> {
  std::vector<float> vals;
  float val;

  while (scanf("%f", &val) == 1) {
    vals.push_back(val);
  }

  return vals;
}

auto main(int argc, char *argv[]) -> int {
  float hext;
  long n_samples;
  long sweeps_per_sample;
  unsigned long seed;
  parse_args(argc, argv, &hext, &n_samples, &sweeps_per_sample, &seed);

  const std::vector<float> temps = read_floats();
  const size_t nt = temps.size();

  int *d_spin;
  float *d_noise;
  float *d_temps;
  unsigned long long *d_naccept;
  int *d_spinsum;
  float *d_m2sum;
  float *d_m4sum;

  hipMalloc(&d_spin, nt * N * sizeof(int));
  hipMalloc(&d_noise, nt * N * sizeof(float));

  hipMalloc(&d_temps, nt * sizeof(float));
  hipMalloc(&d_naccept, nt * sizeof(unsigned long long));
  hipMalloc(&d_spinsum, nt * sizeof(int));
  hipMalloc(&d_m2sum, nt * sizeof(float));
  hipMalloc(&d_m4sum, nt * sizeof(float));

  hipMemcpy(d_temps, temps.data(), nt * sizeof(float), hipMemcpyHostToDevice);

  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);
  hiprandGenerateUniform(gen, d_noise, nt * N);

  k_init_random<<<ceil_div(nt * N, 32), 32>>>(nt, d_noise, d_spin);

  printf("D,L,h_ext,sweeps_per_sample,seed,temperature,sample,accept_rate,"
         "<m^2>,<m^4>,time_s\n");

  for (int isample = 0; isample < n_samples; isample++) {

    hipMemset(d_naccept, 0, nt * sizeof(unsigned long long));

    hipMemset(d_m2sum, 0, nt * sizeof(float));
    hipMemset(d_m4sum, 0, nt * sizeof(float));

    clock_t start_time = clock();

    for (int isweep = 0; isweep < sweeps_per_sample; isweep++) {
      hiprandGenerateUniform(gen, d_noise, nt * N);

      // checkerboard updates

      constexpr dim3 blockDim(32, 1, 1);
      dim3 gridDim(ceil_div(N, blockDim.x), ceil_div(nt, blockDim.z), 1);

      static_assert(blockDim.z == 1, "require blockDim.z == 1");

      k_sweep<<<gridDim, blockDim>>>(
          0, hext, nt, d_temps, d_noise, d_spin, d_naccept); // light squares

      k_sweep<<<gridDim, blockDim>>>(
          1, hext, nt, d_temps, d_noise, d_spin, d_naccept); // dark squares

      // accumulate magnetization

      hipMemset(d_spinsum, 0, nt * sizeof(int));

      k_accum<<<dim3(ceil_div(N, 32), nt, 1), dim3(32, 1, 1)>>>(
          nt, d_spin, d_spinsum);

      k_accum_scalar_moments<<<ceil_div(nt, 32), 32>>>(
          nt, d_spinsum, d_m2sum, d_m4sum);
    }

    std::vector<unsigned long long> naccept(nt);
    hipMemcpy(
        naccept.data(),
        d_naccept,
        nt * sizeof(unsigned long long),
        hipMemcpyDeviceToHost);

    std::vector<float> m2sum(nt), m4sum(nt);
    hipMemcpy(
        m2sum.data(), d_m2sum, nt * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(
        m4sum.data(), d_m4sum, nt * sizeof(float), hipMemcpyDeviceToHost);

    clock_t end_time = clock();

    const double time_s = (double)(end_time - start_time) / CLOCKS_PER_SEC;

    for (int t = 0; t < nt; t++) {
      const double accept_rate =
          (double)naccept[t] / (double)sweeps_per_sample / N;
      const double m2avg = m2sum[t] / (double)sweeps_per_sample;
      const double m4avg = m4sum[t] / (double)sweeps_per_sample;

      printf(
          "%u,%lu,%g,%ld,%ld,%g,%d,%g,%g,%g,%g\n",
          D_,
          L_,
          hext,
          sweeps_per_sample,
          seed,
          temps[t],
          isample,
          accept_rate,
          m2avg,
          m4avg,
          time_s);
    }
  }

  hipFree(d_temps);
  hipFree(d_spin);
  hipFree(d_noise);
  hipFree(d_naccept);
  hipFree(d_spinsum);
  hipFree(d_m2sum);
  hipFree(d_m4sum);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    return 1;
  }

  return 0;
}
