#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand.h>

constexpr unsigned int D_ = 2;

#ifdef L
constexpr unsigned long L_ = L;
#else
constexpr unsigned long L_ = 64;
#endif

constexpr unsigned long N = L_ * L_;

__global__ void k_init_random(
    const float *const __restrict__ noise, int *const __restrict__ spin) {

  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= N)
    return;

  const int p = noise[i] < 0.5;
  spin[i] = 2 * p - 1;
};

__global__ void k_sweep(
    const unsigned int parity,
    const float hext,
    const float temperature,
    const float *__restrict__ noise,
    int *const __restrict__ spin,
    unsigned long long *const __restrict__ naccept) {

  const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= L_ || j >= L_)
    return;

  if ((i + j) % 2 != parity)
    return;

  const unsigned int iprev = (i == 0) ? L_ - 1 : i - 1;
  const unsigned int jprev = (j == 0) ? L_ - 1 : j - 1;

  const unsigned int inext = (i == L_ - 1) ? 0 : i + 1;
  const unsigned int jnext = (j == L_ - 1) ? 0 : j + 1;

  const int nbrsum = spin[i * L_ + jprev] + spin[i * L_ + jnext] +
                     spin[iprev * L_ + j] + spin[inext * L_ + j];

  const float h = static_cast<float>(nbrsum) + hext;
  const unsigned int idx = i * L + j;
  const int s = spin[idx];
  const float de = static_cast<float>(2 * s) * h;

  if (de <= 0) {
    spin[i * L_ + j] = -s;
    atomicAdd(naccept, 1);
  } else {
    const float prob = exp(-de / temperature);
    if (noise[i * L_ + j] < prob) {
      spin[i * L_ + j] = -s;
      atomicAdd(naccept, 1);
    }
  }
}

template <typename T>
__global__ void
k_accum(const T *const __restrict__ vals, T *const __restrict__ out) {

  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= N)
    return;

  atomicAdd(out, vals[i]);
}

constexpr auto ceil_div(const unsigned int x, const unsigned int y)
    -> unsigned int {
  return (x + y - 1) / y;
}

auto parse_float(const char *s) -> float {
  char *endptr = nullptr;
  float r = strtof(s, &endptr);
  if (*endptr != '\0') {
    fprintf(stderr, "Invalid float: %s\n", s);
    exit(1);
  }
  return r;
}

auto parse_long(const char *s) -> long {
  char *endptr = nullptr;
  long r = strtol(s, &endptr, 10);
  if (*endptr != '\0') {
    fprintf(stderr, "Invalid long: %s\n", s);
    exit(1);
  }
  return r;
}

void parse_args(
    int argc,
    char *argv[],
    float *hext,
    long *n_samples,
    long *sweeps_per_sample,
    unsigned long *seed) {
  if (argc != 5) {
    fprintf(
        stderr, "Usage: %s H_EXT N_SAMPLES SWEEPS_PER_SAMPLE SEED\n", argv[0]);
    exit(1);
  }
  *hext = parse_float(argv[1]);
  *n_samples = parse_long(argv[2]);
  *sweeps_per_sample = parse_long(argv[3]);
  *seed = parse_long(argv[4]);
}

auto main(int argc, char *argv[]) -> int {
  float hext;
  long n_samples;
  long sweeps_per_sample;
  unsigned long seed;
  parse_args(argc, argv, &hext, &n_samples, &sweeps_per_sample, &seed);

  int *d_spin;
  float *d_noise;
  unsigned long long *d_naccept;
  int *d_spinsum;

  hipMalloc(&d_spin, N * sizeof(int));
  hipMalloc(&d_noise, N * sizeof(float));
  hipMalloc(&d_naccept, sizeof(unsigned long long));
  hipMalloc(&d_spinsum, sizeof(int));

  printf(
      "D,L,h_ext,sweeps_per_sample,seed,temperature,sample,accept_rate,"
      "<m^2>,<m^4>,time_s\n");

  float temperature;

  while (scanf("%f", &temperature) == 1) {
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandGenerateUniform(gen, d_noise, N);

    k_init_random<<<ceil_div(N, 256), 256>>>(d_noise, d_spin);

    for (int isample = 0; isample < n_samples; ++isample) {
      clock_t start_time = clock();
      double m2sum = 0.0;
      double m4sum = 0.0;

      hipMemset(d_naccept, 0, sizeof(unsigned long long));

      for (int isweep = 0; isweep < sweeps_per_sample; ++isweep) {
        hiprandGenerateUniform(gen, d_noise, N);

        constexpr dim3 blockDim(16, 16);
        dim3 gridDim(ceil_div(L_, blockDim.x), ceil_div(L_, blockDim.y));

        // checkerboard updates
        k_sweep<<<gridDim, blockDim>>>(
            0,
            hext,
            temperature,
            d_noise,
            d_spin,
            d_naccept); // update light squares
        k_sweep<<<gridDim, blockDim>>>(
            1,
            hext,
            temperature,
            d_noise,
            d_spin,
            d_naccept); // update dark squares

        // accumulate magnetization
        hipMemset(d_spinsum, 0, sizeof(int));
        k_accum<<<ceil_div(N, 256), 256>>>(d_spin, d_spinsum);

        int spinsum;
        hipMemcpy(&spinsum, d_spinsum, sizeof(int), hipMemcpyDeviceToHost);
        double m = spinsum / static_cast<double>(N);
        double m2 = m * m;
        double m4 = m2 * m2;
        m2sum += m2;
        m4sum += m4;
      }

      unsigned long long naccept;
      hipMemcpy(
          &naccept,
          d_naccept,
          sizeof(unsigned long long),
          hipMemcpyDeviceToHost);

      clock_t end_time = clock();

      const double accept_rate = static_cast<double>(naccept) /
                                 static_cast<double>(sweeps_per_sample) / N;
      const double m2avg = m2sum / static_cast<double>(sweeps_per_sample);
      const double m4avg = m4sum / static_cast<double>(sweeps_per_sample);
      const double time_s =
          static_cast<double>(end_time - start_time) / CLOCKS_PER_SEC;

      printf(
          "%u,%lu,%g,%ld,%ld,%g,%d,%g,%g,%g,%g\n",
          D_,
          L_,
          hext,
          sweeps_per_sample,
          seed,
          temperature,
          isample,
          accept_rate,
          m2avg,
          m4avg,
          time_s);
    }
  }

  hipFree(d_spin);
  hipFree(d_noise);
  hipFree(d_naccept);
  hipFree(d_spinsum);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    return 1;
  }

  return 0;
}
